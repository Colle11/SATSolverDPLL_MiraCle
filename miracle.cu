#include "hip/hip_runtime.h"
/**
 * miracle.cu: TO-DO.
 * 
 * Copyright (c) Michele Collevati
 */


#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>
#include <math.h>
#include <limits.h>
#include <string.h>


#include "miracle.cuh"
#include "utils.cuh"


/**
 * In branching heuristics, tie breaking by selecting the variable with the
 * smallest index.
 */
#define MIN_IDX_VAR


/**
 * Global variables
 */


static float *lit_weights;      // Array of literal weights.
static int lit_weights_len;     /**
                                 * Length of lit_weights, which is
                                 * mrc->phi->num_vars * 2.
                                 */

static int *clause_sizes;       // Array of clause sizes.
static int clause_sizes_len;    /**
                                 * Length of clause_sizes, which is
                                 * mrc->phi->num_clauses.
                                 */

static int *lit_cnts;           // Array of literal counters.
static int lit_cnts_len;        /**
                                 * Length of lit_cnts, which is
                                 * mrc->phi->num_vars * 2.
                                 */


/**
 * Auxiliary function prototypes
 */


/**
 * @brief Initializes auxiliary data structures.
 * 
 * @param [in]mrc A miracle.
 * @retval None.
 */
static void init_aux_data_structs(Miracle *mrc);


/**
 * @brief Destroys auxiliary data structures.
 * 
 * @retval None.
 */
static void destroy_aux_data_structs();


/**
 * @brief If two_sided = true, computes the JW-TS heuristic,
 * otherwise computes the JW-OS heuristic.
 * 
 * @param [in]mrc A miracle.
 * @param [in]two_sided A flag to choose JW-TS or JW-OS.
 * @retval The branching literal.
 */
static Lit JW_xS_heuristic(Miracle *mrc, bool two_sided);


/**
 * @brief If dlcs = true, computes the DLCS heuristic,
 * otherwise computes the DLIS heuristic.
 * 
 * @param [in]mrc A miracle.
 * @param [in]dlcs A flag to choose DLCS or DLIS.
 * @retval The branching literal.
 */
static Lit DLxS_heuristic(Miracle *mrc, bool dlcs);


/**
 * @brief If rdlcs = true, computes the RDLCS heuristic,
 * otherwise computes the RDLIS heuristic.
 * 
 * @param [in]mrc A miracle.
 * @param [in]rdlcs A flag to choose RDLCS or RDLIS.
 * @retval The branching literal.
 */
static Lit RDLxS_heuristic(Miracle *mrc, bool rdlcs);


/**
 * API definition
 */


Miracle *mrc_create_miracle(char *filename) {
    Miracle *mrc = (Miracle *)malloc(sizeof *mrc);

    mrc->phi = cnf_parse_DIMACS(filename);

    mrc->dec_lvl = 1;

    mrc->var_ass_len = mrc->phi->num_vars;
    mrc->var_ass = (int *)calloc(mrc->var_ass_len,
                                 sizeof *(mrc->var_ass));

    mrc->clause_sat_len = mrc->phi->num_clauses;
    mrc->clause_sat = (int *)calloc(mrc->clause_sat_len,
                                    sizeof *(mrc->clause_sat));

    init_aux_data_structs(mrc);

    return mrc;
}


void mrc_destroy_miracle(Miracle *mrc) {
    cnf_destroy_formula(mrc->phi);
    free(mrc->var_ass);
    free(mrc->clause_sat);
    free(mrc);
    destroy_aux_data_structs();
}


void mrc_print_miracle(Miracle *mrc) {
    printf("*** MiraCle ***\n\n");

    cnf_print_formula(mrc->phi);
    
    printf("Decision level: %d\n", mrc->dec_lvl);

    printf("Variable assignments: ");
    for (int v = 0; v < mrc->var_ass_len; v++) {
        printf("%d ", mrc->var_ass[v]);
    }
    printf("\n");

    printf("Clause satisfiability: ");
    for (int c = 0; c < mrc->clause_sat_len; c++) {
        printf("%d ", mrc->clause_sat[c]);
    }
    printf("\n");

    printf("\n*** End MiraCle ***\n\n");
}


void mrc_assign_lits(Lit *lits, int lits_len, Miracle *mrc) {
    Lit lit;
    Var var;
    bool pol;
    
    // Update variable assignments.
    for (int l = 0; l < lits_len; l++) {
        lit = lits[l];
        var = lit_to_var(lit);
        pol = lit_to_pol(lit);

        mrc->var_ass[var] = pol ? mrc->dec_lvl : -(mrc->dec_lvl);
    }

    Lidx lidx;

    // Update clause satisfiability.
    for (int c = 0; c < mrc->clause_sat_len; c++) {
        if (!(mrc->clause_sat[c])) {
            for (int l = mrc->phi->clause_indices[c];
                 l < mrc->phi->clause_indices[c+1];
                 l++) {
                lidx = mrc->phi->clauses[l];
                var = lidx_to_var(lidx);
                pol = lidx_to_pol(lidx);
                
                if ((pol && mrc->var_ass[var] > 0) ||
                    (!pol && mrc->var_ass[var] < 0)) {
                    mrc->clause_sat[c] = mrc->dec_lvl;
                    break;
                }
            }
        }
    }
}


void mrc_backjump(int bj_dec_lvl, Miracle *mrc) {
    // Restore clause satisfiability.
    for (int c = 0; c < mrc->clause_sat_len; c++) {
        if (mrc->clause_sat[c] > bj_dec_lvl) {
            mrc->clause_sat[c] = 0;
        }
    }

    // Restore variable assignments.
    for (int v = 0; v < mrc->var_ass_len; v++) {
        if (abs(mrc->var_ass[v]) > bj_dec_lvl) {
            mrc->var_ass[v] = 0;
        }
    }

    // Restore decision level.
    mrc->dec_lvl = bj_dec_lvl < 1 ? 1 : bj_dec_lvl;
}


Lit mrc_RAND_heuristic(Miracle *mrc) {
    init_PRNG();

    int num_unass_vars = 0;
    Var bvar = UNDEF_VAR;

    for (Var v = 0; v < mrc->phi->num_vars; v++) {
        if (!(mrc->var_ass[v])) {
            // Variable Selection Heuristic.
            num_unass_vars++;

            if ((rand() % num_unass_vars) == 0) {
                bvar = v;
            }
        }
    }

    if (bvar == UNDEF_VAR) {
        fprintf(stderr, "Undefined variable \"bvar\" in function "
                "\"mrc_RAND_heuristic\".\n");
        exit(EXIT_FAILURE);
    }

    // Polarity Selection Heuristic.
    return rand() % 2 ? varpol_to_lit(bvar, false) : varpol_to_lit(bvar, true);
}


Lit mrc_JW_OS_heuristic(Miracle *mrc) {
    return JW_xS_heuristic(mrc, false);
}


Lit mrc_JW_TS_heuristic(Miracle *mrc) {
    return JW_xS_heuristic(mrc, true);
}


Lit mrc_POSIT_heuristic(Miracle *mrc, const int n) {
    // Clear clause_sizes.
    memset(clause_sizes, 0, sizeof *clause_sizes * clause_sizes_len);

    // Clear lit_cnts.
    memset(lit_cnts, 0, sizeof *lit_cnts * lit_cnts_len);

    int c_size;     // Clause size.
    Lidx lidx;
    Var var;
    int smallest_c_size = INT_MAX;      // Smallest clause size.
    Lidx pos_lidx;
    Lidx neg_lidx;
    int lc_min_pos_lidx;
    int lc_min_neg_lidx;
    int weight;
    int greatest_weight = -1;
    Var bvar = UNDEF_VAR;

    /**
     * Compute the clause sizes and the smallest clause size.
     */
    for (int c = 0; c < mrc->phi->num_clauses; c++) {
        if (!(mrc->clause_sat[c])) {
            c_size = 0;

            for (int l = mrc->phi->clause_indices[c];
                 l < mrc->phi->clause_indices[c+1];
                 l++) {
                lidx = mrc->phi->clauses[l];
                var = lidx_to_var(lidx);

                if (!(mrc->var_ass[var])) {
                    c_size++;
                }
            }

            if (c_size < smallest_c_size) {
                smallest_c_size = c_size;
            }

            clause_sizes[c] = c_size;
        }
    }

    for (int c = 0; c < mrc->phi->num_clauses; c++) {
        if (!(mrc->clause_sat[c]) && clause_sizes[c] == smallest_c_size) {
            for (int l = mrc->phi->clause_indices[c];
                 l < mrc->phi->clause_indices[c+1];
                 l++) {
                lidx = mrc->phi->clauses[l];
                var = lidx_to_var(lidx);

                if (!(mrc->var_ass[var])) {
                    // Variable Selection Heuristic.
                    lit_cnts[lidx]++;
                    
                    pos_lidx = varpol_to_lidx(var, true);
                    neg_lidx = varpol_to_lidx(var, false);
                    lc_min_pos_lidx = lit_cnts[pos_lidx];
                    lc_min_neg_lidx = lit_cnts[neg_lidx];
                    weight = lc_min_pos_lidx * lc_min_neg_lidx *
                             (int)(pow(2, n) + 0.5) +
                             lc_min_pos_lidx + lc_min_neg_lidx;
                    
                    if (weight > greatest_weight
#ifdef MIN_IDX_VAR
                        || ((weight == greatest_weight) && (var < bvar))
#endif
                       ) {
                        bvar = var;
                        greatest_weight = weight;
                    }
                }
            }
        }
    }

    if (bvar == UNDEF_VAR) {
        fprintf(stderr, "Undefined variable \"bvar\" in function "
                "\"mrc_POSIT_heuristic\".\n");
        exit(EXIT_FAILURE);
    }

    pos_lidx = varpol_to_lidx(bvar, true);
    neg_lidx = varpol_to_lidx(bvar, false);
    lc_min_pos_lidx = lit_cnts[pos_lidx];
    lc_min_neg_lidx = lit_cnts[neg_lidx];

    // Polarity Selection Heuristic.
    return lc_min_pos_lidx >= lc_min_neg_lidx ? lidx_to_lit(neg_lidx) :
                                                lidx_to_lit(pos_lidx);
}


Lit mrc_DLIS_heuristic(Miracle *mrc) {
    return DLxS_heuristic(mrc, false);
}


Lit mrc_DLCS_heuristic(Miracle *mrc) {
    return DLxS_heuristic(mrc, true);
}


Lit mrc_RDLIS_heuristic(Miracle *mrc) {
    return RDLxS_heuristic(mrc, false);
}


Lit mrc_RDLCS_heuristic(Miracle *mrc) {
    return RDLxS_heuristic(mrc, true);
}


/**
 * Auxiliary function definitions
 */


static void init_aux_data_structs(Miracle *mrc) {
    lit_weights_len = mrc->phi->num_vars * 2;
    lit_weights = (float *)calloc(lit_weights_len,
                                  sizeof *lit_weights);

    clause_sizes_len = mrc->phi->num_clauses;
    clause_sizes = (int *)calloc(clause_sizes_len,
                                 sizeof *clause_sizes);

    lit_cnts_len = mrc->phi->num_vars * 2;
    lit_cnts = (int *)calloc(lit_cnts_len,
                             sizeof *lit_cnts);
}


static void destroy_aux_data_structs() {
    free(lit_weights);
    free(clause_sizes);
    free(lit_cnts);
}


static Lit JW_xS_heuristic(Miracle *mrc, bool two_sided) {
    // Clear lit_weights.
    memset(lit_weights, 0, sizeof *lit_weights * lit_weights_len);

    int c_size;     // Clause size.
    Lidx lidx;
    Var var;
    Lidx pos_lidx;
    Lidx neg_lidx;
    float weight_pos_lidx;
    float weight_neg_lidx;
    float weight;
    float greatest_weight = -1.0;
    Var bvar = UNDEF_VAR;

    // Compute the JW weight of literals in unresolved clauses.
    for (int c = 0; c < mrc->phi->num_clauses; c++) {
        if (!(mrc->clause_sat[c])) {
            c_size = 0;

            for (int l = mrc->phi->clause_indices[c];
                 l < mrc->phi->clause_indices[c+1];
                 l++) {
                lidx = mrc->phi->clauses[l];
                var = lidx_to_var(lidx);

                if (!(mrc->var_ass[var])) {
                    c_size++;
                }
            }

            for (int l = mrc->phi->clause_indices[c];
                 l < mrc->phi->clause_indices[c+1];
                 l++) {
                lidx = mrc->phi->clauses[l];
                var = lidx_to_var(lidx);

                if (!(mrc->var_ass[var])) {
                    lit_weights[lidx] += powf(2.0, (float)-c_size);
                }
            }
        }
    }

    for (Var v = 0; v < mrc->phi->num_vars; v++) {
        if (!(mrc->var_ass[v])) {
            // Variable Selection Heuristic.
            pos_lidx = varpol_to_lidx(v, true);
            neg_lidx = varpol_to_lidx(v, false);
            weight_pos_lidx = lit_weights[pos_lidx];
            weight_neg_lidx = lit_weights[neg_lidx];
            weight = two_sided ? abs(weight_pos_lidx - weight_neg_lidx) :
                                 (weight_pos_lidx >= weight_neg_lidx ?
                                  weight_pos_lidx : weight_neg_lidx);

            if (weight > greatest_weight) {
                bvar = v;
                greatest_weight = weight;
            }
        }
    }

    if (bvar == UNDEF_VAR) {
        fprintf(stderr, "Undefined variable \"bvar\" in function "
                "\"JW_xS_heuristic\".\n");
        exit(EXIT_FAILURE);
    }

    pos_lidx = varpol_to_lidx(bvar, true);
    neg_lidx = varpol_to_lidx(bvar, false);

    // Polarity Selection Heuristic.
    return lit_weights[pos_lidx] >= lit_weights[neg_lidx] ?
           lidx_to_lit(pos_lidx) : lidx_to_lit(neg_lidx);
}


static Lit DLxS_heuristic(Miracle *mrc, bool dlcs) {
    // Clear lit_cnts.
    memset(lit_cnts, 0, sizeof *lit_cnts * lit_cnts_len);

    Lidx lidx;
    Var var;
    Lidx pos_lidx;
    Lidx neg_lidx;
    int sum;
    int largest_sum = -1;
    Var bvar = UNDEF_VAR;

    for (int c = 0; c < mrc->phi->num_clauses; c++) {
        if (!(mrc->clause_sat[c])) {
            for (int l = mrc->phi->clause_indices[c];
                 l < mrc->phi->clause_indices[c+1];
                 l++) {
                lidx = mrc->phi->clauses[l];
                var = lidx_to_var(lidx);

                if (!(mrc->var_ass[var])) {
                    // Variable Selection Heuristic.
                    lit_cnts[lidx]++;
                    
                    pos_lidx = varpol_to_lidx(var, true);
                    neg_lidx = varpol_to_lidx(var, false);
                    sum = dlcs ? lit_cnts[pos_lidx] + lit_cnts[neg_lidx] :
                                 lit_cnts[lidx];

                    if (sum > largest_sum
#ifdef MIN_IDX_VAR
                        || ((sum == largest_sum) && (var < bvar))
#endif
                       ) {
                        bvar = var;
                        largest_sum = sum;
                    }
                }
            }
        }
    }

    if (bvar == UNDEF_VAR) {
        fprintf(stderr, "Undefined variable \"bvar\" in function "
                "\"DLxS_heuristic\".\n");
        exit(EXIT_FAILURE);
    }

    pos_lidx = varpol_to_lidx(bvar, true);
    neg_lidx = varpol_to_lidx(bvar, false);
    int lc_pos_lidx = lit_cnts[pos_lidx];
    int lc_neg_lidx = lit_cnts[neg_lidx];

    // Polarity Selection Heuristic.
    return lc_pos_lidx >= lc_neg_lidx ? lidx_to_lit(pos_lidx) :
                                        lidx_to_lit(neg_lidx);
}


static Lit RDLxS_heuristic(Miracle *mrc, bool rdlcs) {
    init_PRNG();

    if (rdlcs && (rand() % 2)) {
        return neg_lit(mrc_DLCS_heuristic(mrc));
    } else if (rdlcs) {
        return mrc_DLCS_heuristic(mrc);
    } else if (rand() % 2) {
        return neg_lit(mrc_DLIS_heuristic(mrc));
    } else {
        return mrc_DLIS_heuristic(mrc);
    }
}
